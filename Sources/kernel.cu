#include "hip/hip_runtime.h"
// #include "kernel.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <hiprand.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <algorithm>
#include <vector>


__global__ void initializeCurand(hiprandState * state, const unsigned long int seed, const int mpCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.x * blockIdx.x + threadIdx.y;

	hiprand_init(seed, i*mpCount+j, 0, &state[i*mpCount + j]);
}

__global__ void initializePopulation(hiprandState * state, double* Population, const int NP, const int mpCount)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	// if (i < NP && j < mpCount)
	// {
	// 		Population[i * mpCount + j] = hiprand_uniform(&state[i * mpCount + j]);
	// }
	// for Test
	 if (i < NP && j < mpCount)
	{
			Population[i * mpCount + j] = (i*mpCount+j);
	}


}

__global__ void creatMutationIndexes()
{

}

__global__ void mutateAndCrossOver()
{

}

__global__ void evaluateVasicek()
{

}

__global__ void selectMutatedOrOriginal()
{

}

void runDE()
{
  //set DE Variables
  const int NP = 64;
  const double F = 0.8;
  const double CR = 0.6;

	// Define General Variables
	dim3 threadsPerBlock = (64,3);
	// dim3 numBlocks = (1,1);
  // const int threads = 64; //TODO: think of a good number for blocks and threads
  // const int blocks =  16;

	// Define Generic Variables that later would be changeable
	int mpCount = 3; // This is 3 because CIR and vasicek both have 3 parameters
	std::vector<double> upperBound;
	std::vector<double> lowerBound;
	// for(int i = 0; i<mpCount; ++i)
	// {
	// 	lowerBound[i] = 0.00001;
	// 	upperBound[i] = 0.25;
	// }
	std::cout << "I'm Here" << '\n';
	// Define Host Variables
	thrust::host_vector < double > P(NP * mpCount);

	// Define Device Variables and pointers
	thrust::device_vector < double> dP = P;
	double *dPPointer = thrust::raw_pointer_cast(dP.data());

	// Initialize Curand and genererate the random populations
	hiprandState *dState;
	hipMalloc(&dState, 64 * 3);
	initializeCurand <<< 1,threadsPerBlock >>> (dState , time(NULL), mpCount);
	initializePopulation <<< 1,threadsPerBlock >>> (dState, dPPointer, NP, mpCount);

	P = dP;
	for(int i = 0; i < P.size(); ++i)
		std::cout << "P  in locataion: " << i <<  " is " << P[i] << std::endl;

}

int main()
{
	std::vector<double> mrktData(10,1.0);
	std::cout << "Hi" << '\n';
	runDE();
	return 0;
}
